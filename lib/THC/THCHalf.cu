#include "hip/hip_runtime.h"
#include "THCHalf.h"
#include "THCNumerics.cuh"
#include <thrust/transform.h>
#include <thrust/execution_policy.h>

struct __half2floatOp {
  __device__ float operator()(half v) { return __half2float(v); }
};

struct __float2halfOp {
  __device__ half operator()(float v) { return __float2half(v); }
};

void THCFloat2Half(THCState *state, half *out, float *in, ptrdiff_t len) {
  thrust::transform(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#else
    thrust::device,
#endif
    in, in + len, out, __float2halfOp());
}

void THCHalf2Float(THCState *state, float *out, half *in, ptrdiff_t len) {
  thrust::transform(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#else
    thrust::device,
#endif
    in, in + len, out, __half2floatOp());
}

#if defined (__CUDA_ARCH__) && defined (CUDA_FP16_INSTRINTICS)
template <> const half THCMathTraitsBase<Half>::one() { return THC_FLOAT_TO_HALF(1.); }
template <> const half THCMathTraitsBase<Half>::zero(){ return THC_FLOAT_TO_HALF(0.); }
#endif
